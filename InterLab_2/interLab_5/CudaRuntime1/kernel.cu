﻿#include "hip/hip_runtime.h"
#include ""
#include <locale.h>
#include <stdio.h>
int main() {
	hipDeviceProp_t prop;
	setlocale(LC_ALL, "RUS");
	int count;
	hipGetDeviceCount(&count);
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("-Общая информация об устройстве %d-\n", i);
		printf("Имя: %s\n", prop.name);
		printf("Вычислительные возможности: %d.%d\n",
			prop.major, prop.minor);
		printf("Тактовая частота: %d\n", prop.clockRate);
		printf("Перекрытие копирования: ");
		if (prop.deviceOverlap)
			printf("разрешено\n");
		else printf("запрещено\n");
		printf("Тайм-аут выполнения ядра : ");
		if (prop.kernelExecTimeoutEnabled)
			printf("включен\n");
		else printf("выключен\n");
		printf("-Информация о памяти для устройства %d -\n",
			i);
		printf("Всего глобальной памяти: %ld\n",
			prop.totalGlobalMem);
		printf("Всего константной памяти: %ld\n",
			prop.totalConstMem);
		printf("Максимальный шаг: %ld\n", prop.memPitch);
		printf("Выравнивание текстур: %ld\n",
			prop.textureAlignment);
		printf("Инфо о мультипроцессорах для уст-ва %d -\n",
			i);
		printf("Кол-во мультипроцессоров: %d\n",
			prop.multiProcessorCount);
		printf("Разделяемая память на один МП: %ld\n",
			prop.sharedMemPerBlock);
		printf("Количество регистров на один МП: %d\n",
			prop.regsPerBlock);
		printf("Количество нитей в варпе: %d\n",
			prop.warpSize);
		printf("Макс.кол-во нитей в блоке: %d\n",
			prop.maxThreadsPerBlock);
		printf("Макс.кол-во нитей по измерениям: (%d, %d, % d)\n", prop.maxThreadsDim[0],
			prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
			printf("Максимальные размеры сетки: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
			printf("\n");
	}
	char c = getchar();
}